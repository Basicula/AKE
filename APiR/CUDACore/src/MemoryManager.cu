#include <CUDACore/CUDAUtils.h>
#include <CUDACore/MemoryManager.h>

#include <hip/hip_runtime.h>

void* MemoryManager::operator new(size_t i_len)
  {
  void* ptr;
  CheckCudaErrors(hipMallocManaged(&ptr, i_len));
  CheckCudaErrors(hipDeviceSynchronize());
  return ptr;
  }

void MemoryManager::operator delete(void* ptr) 
  {
  CheckCudaErrors(hipDeviceSynchronize());
  CheckCudaErrors(hipFree(ptr));
  }